#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"

#define checkCudaError(o, l) _checkCudaError(o, l, __func__)
#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)
#include<limits.h>

long long int THREADS_PER_BLOCK = 512;		// This is the numver of threads per block used, and 512 gave the best results
long long int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;	// As each threads takes care of two elements so number of elements is twice of threads

float sequential_scan(long long int* output, long long int* input, long long int length,long long int operation);
float scan(long long int *output, long long int *input, long long int length,long long int operation);
void scanMultiBlock(long long int *output, long long int *input, long long int length,long long int operation,long long int identity);
void scanSingleBlock(long long int *device_output, long long int *device_input, long long int length, long long int operation,long long int identity);
void scanBlockSizedArray(long long int *output, long long int *input, long long int length, long long int operation,long long int identity);
void check(long long int* CPU_Vector,long long int* GPU_Vector, long long int start, long long int end);

__global__ void prescan_SingleBlock(long long int *output, long long int *input, long long int n, long long int nextPowerOfTwo, long long int operation, long long int identity);
__global__ void prescan_MultiBlock(long long int *output, long long int *input, long long int n, long long int* sums, long long int operation,long long int identity);
__global__ void add_two(long long int *output, long long int length, long long int *n1);
__global__ void add_three(long long int *output, long long int length, long long int *n1, long long int *n2);
__global__ void max_two(long long int *output, long long int length, long long int *n1);
__global__ void max_three(long long int *output, long long int length, long long int *n1, long long int *n2);
__global__ void min_two(long long int *output, long long int length, long long int *n1);
__global__ void min_three(long long int *output, long long int length, long long int *n1, long long int *n2);
__global__ void exc_to_inc(long long output, long long input,long long int operations);

void _checkCudaError(const char *message, hipError_t err, const char *caller);
void printResult(const char* prefix, long long int result, float milliseconds);
void printArrayInFile (const char* prefix ,long long int Output[], long long int start, long long int end);



void Scan(long long int N, long long int Option, long long int operation, long long int printing) {
	
	time_t t;
	srand((unsigned)time(&t));
	long long int *in =(long long int *) malloc (sizeof(long long int) * (N+1));	//input array
	long long int *mod_input =(long long int *) malloc (sizeof(long long int) * (N+1));// modified input array incase of Subtraction being the operation
	
	
	if(Option == 1 )		// IF THE ARRAY NEEDS TO BE RANDOMLY GENERATED
	{

		printf("Generating Random Numbers...\n");
		in[0] = rand()%1000000;
		mod_input[0] = in[0];
		
		for (long long int i = 1; i < N; i++) 
		{
			in[i] = rand() % 1000000;
			mod_input[i] = -in[i];	
		}
		if(operation==4)
		{
			in[N] = LONG_LONG_MAX;
		}
		else if (operation==3)
		{
			in[N] = LONG_LONG_MIN;
		}
		else
		{
			in[N] = 0;
			mod_input[N] = 0;	
		
		}

		printf("Finished Generating Random Numbers...\n\n");
	}
	else					// IF THE ARRAY IS FED AS INPUT TO THE PROGRAM
	{
		printf("Please type the desired %lld values of the vector each seperated by an ENTER KEY or WHITESPACE\n",N);
		scanf("%lld",&in[0]);
		mod_input[0] = in[0];
		for (long long int i = 1; i < N; i++) 
		{
			scanf("%lld",&in[i]);
			mod_input[i] = -in[i];
		}
		if(operation==4)
		{
			in[N] = LONG_LONG_MAX;
		}
		else if (operation==3)
		{
			in[N] = LONG_LONG_MIN;
		}
		else
		{
			in[N] = 0;
			mod_input[N] = 0;	
		
		}
		printf("Finished Taking Input...\n\n");
	
	}

	
	
	
	if(printing)		//PRINTING THE OUTPUT ARRAY TO output.txt
	{	
		printf("Printing the Input Vector...\n");
		FILE* fp = fopen("output.txt", "w");		
		printArrayInFile("Original Array",in, 0 , N);
		fclose(fp);
		printf("Finished Printing the Input Vector \n");
	}

	long long int *output_CPU = (long long int *) malloc (sizeof(long long int) * (N+1));
	printf("Doing the sequential Exclusive scan...\n");
	float time_host = sequential_scan(output_CPU, in, (N+1), operation);
	printf("Finished the sequential Exclusive scan...\n\n");
	
	//Printing The Result and Time
	printResult("Host Time  ", output_CPU[N], time_host);
	if(printing)		//PRINTING THE OUTPUT ARRAY TO output.txt
	{	
		printf("Printing the Scanned Vector formed by the CPU...\n");
		printArrayInFile ("HOST RESULT", output_CPU, 1, N+1);
		printf("Finished Printing the Scanned Vector formed by the CPU\n\n");
		
	}
	
	
	// Parallel scan on GPU
	printf("Doing the Parallel Exclusive scan...\n");
	long long int *output_GPU = (long long int *) malloc (sizeof(long long int) * N);
	printf("Finished the Parallel Exclusive scan...\n");
	
	if(operation == 2)		// Special consideration for subtraction because the operation is not associative
	{

		float time_gpu = scan(output_GPU, mod_input, N+1, operation);
		printResult("GPU time ", output_GPU[N], time_gpu);
	}
	else				// for all the other operationators
	{
		float time_gpu = scan(output_GPU, in, N + 1, operation);
		printResult("GPU time ", output_GPU[N], time_gpu);
	}
	if(printing)
	{
		printf("Printing the Scanned Vector formed by the GPU...\n");
		printArrayInFile ("GPU RESULT", output_GPU,1, N + 1);
		printf("Finished Printing the Scanned Vector formed by the GPU\n\n");
		printf("Please look at the output.txt to see the scanned vectors and input vector\n");
	}

	// For checking correctness of solution
	check(output_CPU,output_GPU,1,N + 1);

	//clean up of all memory used up
	free(in);
	free(mod_input);
	//free(output_CPU);
	free(output_GPU);
	}

int main(){
	long long int N=0, options=0, operation=0;
	char printing = 0;
	
	printf("Please input a proper size of the array or vector\n");
	scanf("%lld",&N);
	if(N <= 0)
	{
		printf("Please input a proper number which is greater than zero for the size\n");
		printf("The application would terminate now\n");
		return 0;
	}

	printf("Please select one of the given options \n");
	printf("\t1)Randomize the Elements input array of size %lld\n", N);
	printf("\t2)Proived the Elements of input array of size %lld\n", N);
	printf("Type 1 or 2 depending upon the option you want to select\n");
	scanf("%lld", &options);
	printf("\n");
	if(options!=1&&options!=2)
	{
		printf("Please type either 1 or 2 only next time for selecting the Options\n");
		printf("The application would terminate now\n");
		return 0;
	}

	printf("Please select one of the given operations\n");
	printf("1)Addition\t2)Subtraction\t3)Maximum\t4)Minimum\n");
	printf("Type 1,2,3 or 4 depending upon the operation you want to select\n");
	scanf("%lld", &operation);
	printf("\n");
	if(operation!=1 && operation!=2 && operation!=3 && operation!=4)
	{
		printf("Please type either 1,2,3 OR 4 only next time for selecting the Operator\n");
		printf("The application would terminate now\n");
		return 0;
	}
	getchar();	// to eat the enter key;
	
	printf("Do you wish to print the input and scanned vector in an output.txt file?\n");
	printf("Type y for Yes or n for No\n");
	scanf("%c", &printing);
	printf("\n");

	if(printing!='y'&& printing!='n')
	{
		printf("Please type either character 'y' OR 'n' only next time for choosing to print or not the vectors\n");
		printf("The application would terminate now\n");
		return 0;
	}

	if(printing=='y')
	{
		Scan(N,options,operation,1);
	}
	else
	{
		Scan(N,options,operation,0);
	}
	
	return 0;}

float sequential_scan(long long int* output, long long int* input, long long int length, long long int operation) {
	struct timeval start, end;
    gettimeofday(&start, NULL);
	switch(operation)
	{
		case 1:
		{
			output[0] = 0; // since this is an exclusive scan
			output[1] = input[0]; 
			for (long long int j = 2; j < length; ++j)
			{
				output[j] = input[j - 1] + output[j - 1];
			}
		break;
		}
		case 2:
		{
			output[0] = 0; // since this is an exclusive scan
			output[1] = input[0]; 
			for (long long int j = 2; j < length; ++j)
			{
				output[j] = output[j - 1] - input[j - 1];
			}
		break;
		}
		case 3:
		{
			output[0] = LONG_LONG_MIN;
			output[1] = input[0];
			for (long long int j = 2; j < length; ++j)
			{
				if(input[j-1] > output[j - 1])
					output[j] = input[j-1];
				else
					output[j] = output[j-1];
			}
		break;
		}
		case 4:
		{
			
			output[0] = LONG_LONG_MAX;	//since in exclusive scan the first element is the identity of the operator, in this case the max 								//number possible
			output[1] = input[0];
			for (long long int j = 2; j < length; ++j)
			{
				if(input[j-1] < output[j - 1])
					output[j] = input[j-1];
				else
					output[j] = output[j-1];
			}
		break;
		}
	}
	 
    gettimeofday(&end, NULL);
	float seconds = (end.tv_sec  - start.tv_sec);
    float micros = ((seconds * 1000000)+ (end.tv_usec - start.tv_usec));
	return (float)(micros/1000);}

float scan(long long int *output, long long int *input, long long int length,long long int operation) {
	
	long long int *device_input,*device_output;
	long long int arraySize = length * sizeof(long long int);

	hipMalloc((void **)&device_output, arraySize);
	hipMalloc((void **)&device_input, arraySize);
	hipMemcpy(device_output, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(device_input, input, arraySize, hipMemcpyHostToDevice);

	hipEvent_t initial, final;
	hipEventCreate(&initial);
	hipEventCreate(&final);
	// starting the timer given in CUDA Library
	hipEventRecord(initial);
	long long int identity = 0;
	if(operation==4)
	{
		identity = LONG_LONG_MAX;
	}
	else if (operation==3)
	{
			identity = LONG_LONG_MIN;
	}
	if (length <= ELEMENTS_PER_BLOCK) 
	{
		/*float elapsed = */ scanSingleBlock(device_output, device_input, length, operation, identity);
		//printf("The parallelizable part took %lf ms of time", elapsed);//needed for finding parallelizable part
	}
	else 
	{
		/*float elpased  = */scanMultiBlock(device_output, device_input, length, operation , identity);
		//printf("The parallelizable part took %lf ms of time", elapsed);	//needed for finding parallelizable part
	}

	// end timer
	hipEventRecord(final);
	hipEventSynchronize(final);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, initial, final);

	hipMemcpy(output, device_output, arraySize, hipMemcpyDeviceToHost);

	//clean up
	hipFree(device_input);
	hipFree(device_output);
	hipEventDestroy(final);
	hipEventDestroy(initial);
	

	return elapsedTime;}

/*float*/  //needed return type for finding parallelizable part
void scanMultiBlock(long long int *device_output, long long int *device_input, long long int length,long long int operation, long long int identity) {
	long long int reminder = length % (ELEMENTS_PER_BLOCK);
	
	//float temp_time =0, elapsed_time = 0; //needed for finding parallelizable part
	if (reminder != 0) 
	{
		// perform a large scan on a compatible multiple of elements
		long long int blockMultiple = length - reminder;
		/*elapsed_time += */scanBlockSizedArray(device_output, device_input, blockMultiple,  operation, identity); // needed varaible for finding parallelizable part

		// scan the remaining elements and add the (inclusive) last element of the large scan to this
		long long int *startOfOutputArray = &(device_output[blockMultiple]);
		long long int *startOfInputArray = &(device_input[blockMultiple]);
		/*elapsed_time +=*/ scanSingleBlock(startOfOutputArray, startOfInputArray, reminder,  operation, identity);// needed varaible for finding parallelizable part
		/*
		// needed for finding parallelizable part 
		cudaEvent_t initial, final;
		cudaEventCreate(&initial);
		cudaEventCreate(&final);
		// starting the timer given in CUDA Library
		cudaEventRecord(initial);
		*/
		switch(operation)
		{
			case 1:
			{
				add_three<<<1, reminder>>>(startOfOutputArray, reminder, &(device_input[blockMultiple - 1]), &(device_output[blockMultiple - 1]));
				break;
			}
			case 2:
			{
				add_three<<<1, reminder>>>(startOfOutputArray, reminder, &(device_input[blockMultiple - 1]), &(device_output[blockMultiple - 1]));
				break;
			}
			case 3:
			{
				max_three<<<1, reminder>>>(startOfOutputArray, reminder, &(device_input[blockMultiple - 1]), &(device_output[blockMultiple - 1]));
				break;
			}
			
			case 4:
			{
				min_three<<<1, reminder>>>(startOfOutputArray, reminder, &(device_input[blockMultiple - 1]), &(device_output[blockMultiple - 1]));
				break;
			}
		}
		/*
		Needed for finding parallelizable part
		cudaEventRecord(final);
		cudaEventSynchronize(final);
		cudaEventElapsedTime(&temp_time, initial, final);
		elapsed_time += temp_time
		cudaFree(device_input);
		cudaFree(device_output);
		cudaEventDestroy(final);
		cudaEventDestroy(initial);
		return elapsed_time;
		*/

	}
	else 
	{
		// Both the comments are needed for finding parallelizable part 
		/*float elapsed_time = */ scanBlockSizedArray(device_output, device_input, length,  operation, identity);	
		/*return elapsed_time;*/
	}
}

/*float*/
void scanSingleBlock(long long int *device_output, long long int *device_input, long long int length,long long int operation, long long int identity) {

		long long int nextPowerOfTwo = 1;
		while (nextPowerOfTwo < length) 
		{
			nextPowerOfTwo *= 2;
		}
		/*
		Needed for finding parallelizable part
		float elapsed_time = 0;
		cudaEvent_t initial, final;
		cudaEventCreate(&initial);
		cudaEventCreate(&final);
		// starting the timer given in CUDA Library
		cudaEventRecord(initial);
		*/
		prescan_SingleBlock<<<1, (length + 1) / 2, 2 * nextPowerOfTwo * sizeof(long long int)>>>(device_output, device_input, length, nextPowerOfTwo, operation, identity);	
		/*
		Needed for finding parallelizable part
		
		cudaEventRecord(final);
		cudaEventSynchronize(final);
		cudaEventElapsedTime(&elapsed_time, initial, final);
		cudaFree(device_input);
		cudaFree(device_output);
		cudaEventDestroy(final);
		cudaEventDestroy(initial);
		return elapsed_time;
		*/
}

/*float*/
void scanBlockSizedArray(long long int *device_output, long long int *device_input, long long int length, long long int operation, long long int identity) {
	long long int num_blocks = length / ELEMENTS_PER_BLOCK;
	long long int sharedMemBlockSize = ELEMENTS_PER_BLOCK * sizeof(long long int);

	long long int *device_blocks, *device_inputcr;
	hipMalloc((void **)&device_blocks, num_blocks * sizeof(long long int));
	hipMalloc((void **)&device_inputcr, num_blocks * sizeof(long long int));
	/*
	Needed for finding parallelizable part	
	float elapsed_time = 0, temp_time;
	cudaEvent_t initial, final;
	cudaEventCreate(&initial);
	cudaEventCreate(&final);
	// starting the timer given in CUDA Library
	cudaEventRecord(initial);
	*/
	prescan_MultiBlock<<<num_blocks, THREADS_PER_BLOCK, 2*sharedMemBlockSize>>>(device_output, device_input, ELEMENTS_PER_BLOCK, device_blocks, operation, identity);
	/*
	Needed for finding parallelizable part

	cudaEventRecord(final);
	cudaEventSynchronize(final);
	cudaEventElapsedTime(&elapsed_time, initial, final);
	cudaFree(device_input);
	cudaFree(device_output);
	cudaEventDestroy(final);
	cudaEventDestroy(initial);
	*/
	if ((num_blocks + 1) / 2 < THREADS_PER_BLOCK) 
	{
		//Needed for finding parallelizable part
		/*elapsed_time+=*/scanSingleBlock(device_inputcr, device_blocks, num_blocks,  operation, identity);
	}
	else 
	{
		// Needed for finding parallelizable part
		/*elapsed_time+= */scanMultiBlock(device_inputcr, device_blocks, num_blocks,  operation, identity);
	}
	/*
	Needed for finding parallelizable part
	cudaEventCreate(&initial);
	cudaEventCreate(&final);
	// starting the timer given in CUDA Library
	cudaEventRecord(initial);
	*/
	switch(operation)
		{
			case 1:
			{
				add_two<<<num_blocks, ELEMENTS_PER_BLOCK>>>(device_output, ELEMENTS_PER_BLOCK, device_inputcr);
				break;
			}
			case 2:
			{
				add_two<<<num_blocks, ELEMENTS_PER_BLOCK>>>(device_output, ELEMENTS_PER_BLOCK, device_inputcr);
				break;
			}
			case 3:
			{
				max_two<<<num_blocks, ELEMENTS_PER_BLOCK>>>(device_output, ELEMENTS_PER_BLOCK, device_inputcr);
				break;
			}
			
			case 4:
			{
				min_two<<<num_blocks, ELEMENTS_PER_BLOCK>>>(device_output, ELEMENTS_PER_BLOCK, device_inputcr);
				break;
			}
		}
	/*
	Needed for finding parallelizable part
	
	cudaEventRecord(final);
	cudaEventSynchronize(final);
	cudaEventElapsedTime(&temp_time, initial, final);
	cudaFree(device_input);
	cudaFree(device_output);
	cudaEventDestroy(final);
	cudaEventDestroy(initial);
	elapsed_time +=  temp_time
	cudaFree(device_inputcr);
	cudaFree(device_blocks);
	return elapsed_time;
	*/
	}

__global__ void prescan_SingleBlock(long long int *output, long long int *input, long long int n, long long int nextPowerOfTwo, long long int operation, long long int identity)
{
	extern __shared__ long long int temp[];
	long long int threadID = threadIdx.x;
	long long int offset = 1;

	long long int index1 = threadID;
	long long int index2 = threadID + (n / 2);
	long long int bankOffsetB = CONFLICT_FREE_OFFSET(index2);
	long long int bankOffsetA = CONFLICT_FREE_OFFSET(index1);

	if (threadID >= n) {
		temp[index1 + bankOffsetA] = 0;
		temp[index2 + bankOffsetB] = 0;
	}

	else 
	{
			temp[index1 + bankOffsetA] = input[index1];
			temp[index2 + bankOffsetB] = input[index2];
	}


	for (long long int d = nextPowerOfTwo/2; d > 0; d= d/2) // Do the reduction by building a operation(like sum) tree in place
	{
		__syncthreads();
		
		if (threadID < d)
		{
			long long int index1 = offset * (2 * threadID + 1) - 1;
			long long int index2 = offset * (2 * threadID + 2) - 1;
			index1 += CONFLICT_FREE_OFFSET(index1);
			index2 += CONFLICT_FREE_OFFSET(index2);
			switch(operation)
			{
				case 1:
				{
					temp[index2] += temp[index1];
					break;
				}
			
				case 2:
				{
					temp[index2] += temp[index1];
					break;
				}
				case 3:
				{
					if(temp[index2] < temp[index1])
						temp[index2] = temp[index1];
					break;
				}
				case 4:
				{
					if(temp[index2] > temp[index1])
						temp[index2] = temp[index1];
					break;
				}}
		}
		offset *= 2;
	}
	__syncthreads();
	
	if (threadID == 0) 
	{		/*
			//FOR DEBUGGIN PURPOSE PLEASE IGNORE
			printf("%s\n", "TEMP");
			for(long long int i=0; i < n; i++)
			{
				printf("%lld ",temp[i]);
			}
			printf("\n");
			*/
		if(operation!=4)
		{
				temp[nextPowerOfTwo - 1 + CONFLICT_FREE_OFFSET(nextPowerOfTwo - 1)] = 0; // clear the last element for exclusive scan
		}
		else
		{
				temp[nextPowerOfTwo - 1 + CONFLICT_FREE_OFFSET(nextPowerOfTwo - 1)] = identity; // clear the last element for exclusive scan
		}
	}

	for (long long int d = 1; d < nextPowerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset = offset/2;
		__syncthreads();
		if (threadID < d)
		{
			long long int index1 = offset * (2 * threadID + 1) - 1;
			long long int index2 = offset * (2 * threadID + 2) - 1;
			index1 += CONFLICT_FREE_OFFSET(index1);
			index2 += CONFLICT_FREE_OFFSET(index2);

			long long int t = temp[index1];
			temp[index1] = temp[index2];
			switch(operation)
			{
				case 1:
				{
					temp[index2] += t;
					break;
				}
				case 2:
				{
					temp[index2] += t;
					break;
				}
				case 3:
				{
					if(temp[index2] < t)
						temp[index2] = t;
					break;
				}
				case 4:
				{
					if(temp[index2] > t)
						temp[index2] = t;
					break;
				}}
		}
	}
	__syncthreads();

	if (threadID < n) 
	{
		output[index1] = temp[index1 + bankOffsetA];
		output[index2] = temp[index2 + bankOffsetB];
	}}

__global__ void prescan_MultiBlock(long long int *output, long long int *input, long long int n, long long int *sums,long long int operation,long long int identity) {
	extern __shared__ long long int temp[];

	long long int blockID = blockIdx.x;
	long long int threadID = threadIdx.x;
	long long int blockOffset = blockID * n;

	long long int index1 = threadID;
	long long int index2 = threadID + (n / 2);
	long long int bankOffsetA = CONFLICT_FREE_OFFSET(index1);
	long long int bankOffsetB = CONFLICT_FREE_OFFSET(index2);
	temp[index1 + bankOffsetA] = input[blockOffset + index1];
	temp[index2 + bankOffsetB] = input[blockOffset + index2];

	long long int offset = 1;
	for (long long int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			long long int index1 = offset * (2 * threadID + 1) - 1;
			long long int index2 = offset * (2 * threadID + 2) - 1;
			index1 += CONFLICT_FREE_OFFSET(index1);
			index2 += CONFLICT_FREE_OFFSET(index2);
			switch(operation)
			{
				case 1:
				{
					temp[index2] += temp[index1];
					break;
				}
				case 2:
				{
					temp[index2] += temp[index1];
					break;
				}
				case 3:
				{
					if(temp[index2] < temp[index1])
						temp[index2] = temp[index1];
					break;
				}
				case 4:
				{
					if(temp[index2] > temp[index1])
						temp[index2] = temp[index1];
				}
			}
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) 
	{
		sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
		if(operation!=4)
		{
			temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
		}
		else
		{
			temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = identity;
		}
	}

	for (long long int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			long long int index1 = offset * (2 * threadID + 1) - 1;
			long long int index2 = offset * (2 * threadID + 2) - 1;
			index1 += CONFLICT_FREE_OFFSET(index1);
			index2 += CONFLICT_FREE_OFFSET(index2);

			long long int t = temp[index1];
			temp[index1] = temp[index2];
			switch(operation)
			{
				case 1:
				{
					temp[index2] += t;
					break;
				}
				case 2:
				{
					temp[index2] += t;
					break;
				}
				case 3:
				{
					if(temp[index2] < t)
						temp[index2] = t;
					break;
				}
				case 4:
				{
					if(temp[index2] > t)
						temp[index2] = t;
					break;
				}
			}
		}
	}
	__syncthreads();

	output[blockOffset + index1] = temp[index1 + bankOffsetA];
	output[blockOffset + index2] = temp[index2 + bankOffsetB];
}


//functions to add two or three numbers in given arrays
__global__ void add_two(long long int *output, long long int length, long long int *n) {
	long long int blockID = blockIdx.x;
	long long int threadID = threadIdx.x;
	long long int blockOffset = blockID * length;

	output[blockOffset + threadID] += n[blockID];}
__global__ void add_three(long long int *output, long long int length, long long int *n1, long long int *n2) {
	long long int blockID = blockIdx.x;
	long long int threadID = threadIdx.x;
	long long int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];}

//functions to find maximum of two or three numbers in given arrays
__global__ void max_two(long long int *output, long long int length, long long int *n) {
	long long int blockID = blockIdx.x;
	long long int threadID = threadIdx.x;
	long long int blockOffset = blockID * length;
	if(output[blockOffset + threadID] < n[blockID])
	{
			output[blockOffset + threadID] = n[blockID];
	}}
__global__ void max_three(long long int *output, long long int length, long long int *n1, long long int *n2) {
	long long int blockID = blockIdx.x;
	long long int threadID = threadIdx.x;
	long long int blockOffset = blockID * length;
	if(n1[blockID] > n2[blockID])
	{
		if(n1[blockID] > output[blockOffset + threadID])
		{
			output[blockOffset + threadID] = n1[blockID];
		}
	}
	else
	{
		if(n2[blockID] > output[blockOffset + threadID])
		{
			output[blockOffset + threadID] = n2[blockID];
		}
	}}

//functions to find minimum of two or three numbers in given arrays
__global__ void min_two(long long int *output, long long int length, long long int *n) {
	long long int blockID = blockIdx.x;
	long long int threadID = threadIdx.x;
	long long int blockOffset = blockID * length;

	if(output[blockOffset + threadID] > n[blockID])
	{
			output[blockOffset + threadID] = n[blockID];
	}}
__global__ void min_three(long long int *output, long long int length, long long int *n1, long long int *n2) {
	long long int blockID = blockIdx.x;
	long long int threadID = threadIdx.x;
	long long int blockOffset = blockID * length;

	if(n1[blockID] < n2[blockID])
	{
		if(n1[blockID] < output[blockOffset + threadID])
		{
			output[blockOffset + threadID] = n1[blockID];
		}
	}
	else
	{
		if(n2[blockID] < output[blockOffset + threadID])
		{
			output[blockOffset + threadID] = n2[blockID];
		}
	}}

void _checkCudaError(const char *message, hipError_t err, const char *caller) {
	if (err != hipSuccess) {
		fprintf(stderr, "Error in: %s\n", caller);
		fprintf(stderr, "%s\n", message);
		fprintf(stderr, ": %s\n", hipGetErrorString(err));
		exit(0);}}

void printResult(const char* Heading, long long int result, float milliseconds) {
	printf("%s\n", Heading);
	printf("Final Reduction is %lld and it was done in %lf ms\n", result, milliseconds);}

void printArrayInFile (const char* Header ,long long int Output[], long long int start, long long int end){
	FILE* fp = fopen("output.txt", "a");
	fprintf(fp, "%s\n", Header);
	for(long long int i=start; i < end; i++)
	{
		fprintf(fp, "%lld ",Output[i]);
	}
	fprintf(fp, "\n");
	fclose(fp);}

void check(long long int* CPU_Vector,long long int* GPU_Vector, long long int start, long long int end)
{
	for(int i=start; i < end; i++)
	{
		if(CPU_Vector[i]!=GPU_Vector[i])
		{
			printf("Outputs don't match\n");
			return;
		}
	}
	printf("Outputs do match, The implementation is successful\n");}
